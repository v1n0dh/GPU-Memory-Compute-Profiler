#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void fma_kernel(float* a, float* b, float* c, size_t n, int iters) {
    size_t i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= n) return;
    float x = a[i];
    float y = b[i];
    float z = c[i];
    #pragma unroll 4
    for (int k = 0; k < iters; ++k) {
        // 2 FLOPs per FMA
        z = fmaf(x, y, z);
        x = fmaf(y, z, x);
        y = fmaf(z, x, y);
    }
    c[i] = z;
}

extern "C" hipError_t launch_fma_kernel(float* a, float* b, float* c,
                                         size_t n, int iters, float* ms_out) {
    hipError_t err;
    int block = 256;
    int grid = static_cast<int>((n + block - 1) / block);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    err = hipDeviceSynchronize(); if (err) return err;
    hipEventRecord(start);
    fma_kernel<<<grid, block>>>(a, b, c, n, iters);
    hipEventRecord(stop);
    err = hipGetLastError(); if (err) return err;
    hipEventSynchronize(stop);
    hipEventElapsedTime(ms_out, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return hipSuccess;
}

